#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include "utils.h"
#include "accessMode.cuh"
#include "expand.cuh"
#include <hip/hip_runtime.h>

using namespace std;
void expand(int i) {
	return ;
}
__global__ void set_validation(OffsetT *row_start, uint8_t *valid_candi, uint32_t nnodes, uint32_t min_deg) {
	uint32_t tid = threadIdx.x + blockDim.x*blockIdx.x;
	for (uint32_t i = tid; i < nnodes; i += (blockDim.x*gridDim.x)) {
		if (row_start[i+1] - row_start[i] >= min_deg)
			valid_candi[i] = 1;
	}
	return ;
}
int main(int argc, char *argv[]) {
	if (argc < 4) {
		printf("usage: ./kcl ($filename) ($clique size) graph_mem_type debug\n");
		return 0;
	}
	if (string(argv[argc-1]) != "debug") {
		log_set_quiet(true);
	}
	Clock start("Start");
	uint32_t k = std::atoi(argv[2]);
	assert(k <= embedding_max_length);
	std::string file_name = argv[1];
	CSRGraph data_graph;
	mem_type mt_emb = (mem_type)1;//0 GPU 1 Unified 2 Zero 3 Combine
	mem_type mt_graph = (mem_type)atoi(argv[3]);
	if (mt_graph > 1)
		check_cuda_error(hipSetDeviceFlags(hipDeviceMapHost));
	data_graph.read(file_name, false, mt_graph);//no label for k-clique
	log_info(start.start());
	log_info(start.count("nedges %lu, nnodes %d", data_graph.get_nedges(), data_graph.get_nnodes()));
	EmbeddingList emb_list;
	uint32_t nnodes = data_graph.get_nnodes();
	uint64_t nedges = data_graph.get_nedges();
	log_info(start.count("embedding initialization done!"));
	//check_cuda_error(hipDeviceSynchronize());
	//TODO: here we plan to add a optimizer to determine expand order, expand constraint, and so on.
	//set the first level
	KeyT *seq, *results; 
	hipMalloc((void **)&seq, sizeof(KeyT)*nnodes);
	check_cuda_error(hipMalloc((void **)&results, sizeof(KeyT)*nnodes));
	check_cuda_error(hipMemset(results, -1, sizeof(KeyT)*nnodes));
	uint8_t *valid_candi;
	check_cuda_error(hipMalloc((void **)&valid_candi, sizeof(uint8_t)*nnodes));
	check_cuda_error(hipMemset(valid_candi, 0, sizeof(uint8_t)*nnodes));
	set_validation<<<10000, 256>>>(data_graph.row_start, valid_candi, nnodes, k-1);
	thrust::sequence(thrust::device, seq, seq + nnodes);
	uint32_t valid_node_num = thrust::copy_if(thrust::device, seq, seq + nnodes, valid_candi, results, is_valid())- results;
	check_cuda_error(hipDeviceSynchronize());
	emb_list.init(valid_node_num, k, mt_emb, false);
	emb_list.copy_to_level(0, results, 0, valid_node_num);
	check_cuda_error(hipFree(seq));
	check_cuda_error(hipFree(results));

	//set the second level
	//emb_list.add_level(nedges);
	//expand for every vertex in the query graph
	access_mode_controller access_controller;
	access_controller.set_vertex_page_border(data_graph);
	log_info(start.count("access controller initalization done!"));
	Clock Expand("Expand");
	log_info(Expand.start());
	for (int i = 1; i < k; i ++) {
		//construct the expand constraint
		uint64_t _nbrs = 0, _order_nbr = 0;
		//int8_t *_order_nbr_cmp = new int8_t [i];
		for (uint8_t j = 0; j < i; j ++) {
			_nbrs = _nbrs | (j << (j*8));
			//_order_nbr_cmp[j] = 1;
			_order_nbr = _order_nbr | (j << (j*8));
		}
		expand_constraint ec((node_data_type)0xff, (uint8_t)k-1, _nbrs, (uint8_t)i, 
							 (emb_order)1, _order_nbr, (uint8_t)i);
		//expand
		log_info(Expand.count("for the %dth iteration, start expand... ...",i));
		bool write_back = i == k-1 ? false : true;
		expand_dynamic(data_graph, emb_list, i, ec, write_back);
		//expand_in_batch(data_graph, emb_list, i, ec);
		log_info(Expand.count("for the %dth iteration, end expand",i));
		Expand.pause();
		//emb_off_type results = emb_list.check_valid_num(i);
		Expand.goon();
		//set access mode
		if (mt_graph == 3) {
			Expand.pause();
			access_controller.cal_access_mode_by_EL(data_graph, ec, emb_list);
			Expand.goon();
		}
		log_info(Expand.count("for the %dth iteration, end set access mode",i));
		//delete ec;
	}
	log_info(Expand.count("end expand"));
	log_info(start.count("k-clique count ends."));
	//#TODO copy the results back to CPU and check the results;
	//CSRGraph data_graph_h;
	//data_graph.copy_to_cpu(data_graph_h);
	//#show the results in data_graph_h
	emb_list.clean();
	access_controller.clean();
	data_graph.clean();

	return 0;
}
	
